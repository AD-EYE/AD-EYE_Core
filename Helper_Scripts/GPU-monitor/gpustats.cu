#include <iostream>
#include <iomanip>
#include <unistd.h>
#include "hip/hip_runtime.h"

int main()
{
    size_t free_byte ;
    size_t total_byte ;
    int device ;
    int nDevices;
    hipDeviceProp_t prop;

    // Get and print GPU information for all precent devices
    hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
            hipGetDeviceProperties(&prop, i);
            std::cout << "\nDevice Number: " << i << "\nDevice name: " << prop.name << "\nMemory Clock Rate (KHz): " << prop.memoryClockRate << "\nMemory Bus Width (bits): " << "\nMemory Size (MB): " << prop.totalGlobalMem/1000000 << prop.memoryBusWidth << "\nPeak Memory Bandwidth (GB/s): " << 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6 << "\n" << std::endl;
	}
 
    std::cout << "\n" << std::endl;

    while (true)
    {
        hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
	
        if ( hipSuccess != cuda_status ){
            std::cout << "Error: hipMemGetInfo fails, " << hipGetErrorString(cuda_status) << std::endl;
            exit(1);
        }
        
        //Get the currently used GPU device ID
	    cuda_status = hipGetDevice( &device );

	    if ( hipSuccess != cuda_status ){
            std::cout << "Error: hipGetDevice fails, " << hipGetErrorString(cuda_status) << std::endl;
            exit(1);
        }

        //Calculate the memory usage
        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;
    
        //Set the decimal precision
        std::cout << std::fixed;
	    std::cout << std::setprecision(2); // 2 = 2 decimal points
        
        //Print the GPU id and Memory usage in percentage
        std::cout << "GPU STATS FOR DEVICE " << device << std::endl;	
	    std::cout << "Memory used: " << (used_db/total_db)*100 << " %\n" << std::endl;
        
        //Sleep 1 second befor sampling again 
        sleep(1);
    }

    //return 0;
}
